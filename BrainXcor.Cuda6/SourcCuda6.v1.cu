#include "hip/hip_runtime.h"

#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>
#include "stdio.h"
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>  

#define THREADS 512
//#define Gy 32 //grid y dimension
//#define Gx 1920 //grid x dimension
//#define Xt 1 //thread x dimension
//#define Yt 64 //thread y dimension
//#define Fx 2000 //number of frames
typedef struct{
	int x;
	int y;
} twoD;


typedef struct{
	unsigned int win;
	float sDev;
} pixelLoc;

typedef struct{
	unsigned int loc_Wind1;
	unsigned int loc_Wind2;
	float loc_corrCoef;
} PixelxCor;

int const Gy = 32; //grid y dimension
int const Gx = 4; //grid x dimension
int const Xt = 512; //thread x dimension
//int const Gx = 1920; //grid x dimension
//int const Xt = 1; //thread x dimension
int const Yt = 1; //32//thread y dimension
int const Fx = 2000; //number of frames
int const h_Wsize = 50;


using namespace std;

//this where each thread takes a different window to Xcorr
//__global__ void XcrossCUDA(int* d_Pixels, pixelLoc* d_PL, float* d_Cor, int N, int corCount, int Wsize)
//{
//	int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
//	//	int yIdx = blockIdx.y * blockDim.y + threadIdx.y;
//
//	float x1, x2, SumPt2, Sum_X1X2, window[120],sdev1,sdev2;
//	int winStart, window1,window2, Index,temp2,temp3; //change yIdx and xIdx
//	//	float xcorrValues[16000];
//
//	// find local point only for xcorr with window
//	if(xIdx < N-1 && d_PL[xIdx].win > 0)//|| window1 > 0) //not at end of file
//	{
//		winStart = xIdx;
//		temp3 = xIdx; //index from "points to correlate" array d_PL
//		window1 = d_PL[winStart].win;
//		//load window for xcorr
//		sdev1 = d_PL[winStart].sDev;
//
//		//temp2 = ((temp3) * (temp3-1))/2;
//		//Index = (N * temp3) - temp2; //this needs to be checked
//		//calculate the offset to write the output data file [N * I - (N(N-1)/2)]
//		//if(xIdx == 0)
//		//	Index = 0;
//		//else 
//		//	{
//		//	temp2 = ;
//		Index = corCount - ((N-xIdx) * (N-xIdx - 1))/2; //this needs to be checked
//		//	}
//
//		//get pixel values for correlation's Master window
//		for(int n = 0; n < Wsize; n++) 
//			window[n] = d_Pixels[window1 + n];
//
//		//now get windows for other points in the correlation
//		for(int i = 0; i < N - xIdx; i++)
//		{
//			window2 = d_PL[winStart+i].win;
//			sdev2 = d_PL[winStart+i].sDev;
//			//			if(sdev2 == 0 || sdev1 == 0) 
//			//			continue;
//			//if(window2 < 0)
//			//	break;
//			//find data start point for windows in silo
//
//			//if point is valid then begin correlations
//			x1 = x2 = Sum_X1X2 = 0.;
//
//			// do the actual cross correlation now
//			for (int l = 0; l < Wsize; l++)
//			{
//				SumPt2 = d_Pixels[window2 + l];
//				x1 += window[l];
//				x2 += SumPt2;
//				Sum_X1X2 += window[l] * SumPt2;
//			}
//			d_Cor[i + Index] = ((Sum_X1X2 - x1 * x2/Wsize)/(Wsize - 1)/sdev2/sdev1);	
//		} //end of correlation calculation
//	} // end of finding windows inside pixel silo
//}

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) 
	{
		cerr<<hipGetErrorString( err )<<" in "<<file<<" at line "<<line<<endl;
		//		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}

//this where thread acts on the same window to Xcorr
__global__ void XcrossCUDA_same(int* d_Pixels, pixelLoc* d_PL, PixelxCor* d_Cor, int X, int corCount, int Wsize)
{
	extern __shared__ int window[];
	//here d_Cor is on Host not Device
	unsigned int xIdx = blockIdx.x * blockDim.x + threadIdx.x;

	float x1, x2, SumPt2, Sum_X1X2,sdev1,sdev2;
	unsigned int winStart, window1,window2, Index; //change yIdx and xIdx

	// find local point only for xcorr with window
	if(xIdx < X-1)
	{
		for(int i = 0, j = xIdx; xIdx < X-1 - i;j = xIdx, i++) //increment through all PL data points
		{
			__syncthreads();  //need this so wndow not changed while still in use.
			Index = corCount - ((X-i) * (X-i - 1))/2; //this needs to be checked
			winStart = i; //index of the window
			window1 = d_PL[winStart].win;
			sdev1 = d_PL[winStart].sDev;
			//get pixel values for correlation's Master window
			//MUST use threadIdx not xIdx window must exists in each block.
			if(threadIdx.x == 0)
			{
				for(int ii = 0; ii < Wsize; ii++)
				{
					window[ii] = d_Pixels[window1 + ii]; // check this...
				}
			}
			__syncthreads();

			//roll through all the data for this window
			while(j < X-1-i)
			{
				window2 = d_PL[winStart+j].win;
				sdev2 = d_PL[winStart+j].sDev;

				//if point is valid then begin correlations
				x1 = x2 = Sum_X1X2 = 0.;

				// do the actual cross correlation now
				for (int l = 0; l < Wsize; l++)
				{
					SumPt2 = d_Pixels[window2 + l];
					x1 += window[l];
					x2 += SumPt2;
					Sum_X1X2 += window[l] * SumPt2;
				}
				//if(x14 > 1)
				//	x14 = x14;
				//				((Sum_X1X2 - x1 * x2/Wsize)/(Wsize - 1)/sdev1/sdev2);	
				d_Cor[j + Index].loc_corrCoef = ((Sum_X1X2 - x1 * x2/Wsize)/(Wsize - 1)/sdev1/sdev2);
				d_Cor[j + Index].loc_Wind1 = window1;	
				d_Cor[j + Index].loc_Wind2 = window2;	

				j += gridDim.x * blockDim.x;
			}
		}
	}
}


__global__ void StdDev(int* d_Pixels, pixelLoc* d_PL,  int Wsize, int frames,  int yTotal, twoD numProcThds, int devThres)
{
	unsigned int xIdx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int yIdx = blockIdx.y * blockDim.y + threadIdx.y;

	float temp, x1=0.f, x2=0.f;
	unsigned int xyStart; //where to start reading the window
	unsigned int outStart;   //output file indexing
	if(xIdx < numProcThds.x && yIdx < numProcThds.y)
	{
		while(yIdx < yTotal)
		{
			//Calculate Standard Deviation per window within XY pixel silo
			xyStart = xIdx + frames * yIdx;
			outStart = xIdx + numProcThds.x * yIdx;
			x1 = x2 = 0.;
			for(int i = 0; i < Wsize; i++)
			{
				temp = (float)d_Pixels[xyStart + i];
				x1 += temp;
				x2 += temp * temp; 
			}
			temp = sqrtf((x2 - x1*x1/Wsize)/(Wsize-1));
			d_PL[outStart].win = xyStart;
			if(temp > devThres)
				d_PL[outStart].sDev = temp;
			else
				d_PL[outStart].sDev = 0.0f;
			yIdx += gridDim.y*blockDim.y;
		}
	}
}


int main()
{
	twoD numProcThds;
	numProcThds.x = Fx - h_Wsize; //used in Stdev kernel for total number threads X direction
	numProcThds.y = Gy*Yt;//used in Stdev kernel for total number threads Y direction

	int const count = Fx*Gy*Yt; //Fx=MaxX, Gy * Yt = maxY for data file
	int const imageX = 172;  //size of Image used ... columns
	int const imageY = 130; //size of Image used ... rows
	int const totalPixs = imageX * imageY; //total pixel number for image
	int const readSize = Fx * totalPixs; //total memory size of all data
	int devThres = 35;
	int procsrTot = numProcThds.x*numProcThds.y;
	int Xloc1, Yloc1, Floc1; //used for X,Y,Frame for Point 1
	int Xloc2, Yloc2, Floc2; //used for X,Y,Frame for Point 2
	int deviceCount;
	int *d_Pixels;   //device version of h_Pixels
	int *h_Pixels = new int[readSize];  //used to hold pixel values
	int i = 0, N;
	int size_file=0;
	int abc = sizeof(int);
	int  asd=sizeof(pixelLoc);

	pixelLoc *d_PL; //device version of h_PL
	pixelLoc *PL,*h_PL = new pixelLoc[readSize];  //used to hold Stdev values
	PixelxCor *d_Cor;  //device version of h_Cor
	hipError_t  code;

	//this MUST be here; flags must be set before any
	//Cuda calls made; if Host Memory use by Device is used!!
	hipSetDeviceFlags(hipDeviceMapHost);

	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) 
	{
		fprintf(stderr, "error: no devices supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}

	int dev = 0;
	hipSetDevice(dev);
	int *Pixels;
	hipDeviceProp_t devProps;


	if (hipGetDeviceProperties(&devProps, dev) == 0)
	{
		printf("Using device %d:\n", dev);
		printf("%s; global mem: %uB; compute v%d.%d; clock: %d kHz\n",
			devProps.name, (long)devProps.totalGlobalMem, 
			(int)devProps.major, (int)devProps.minor, 
			(int)devProps.clockRate);
	}
	int const gridLimit = devProps.maxGridSize[0];
	int  thredMax = devProps.maxThreadsPerBlock; //devProps.maxThreadsPerBlock;

	const dim3 blockSize(Xt, Yt, 1);  //TODO
	const dim3 gridSize(Gx,Gy, 1);  //TODO
	//int Tot_NumThreads;
	//int BlockWidth;
	//first = Tot_NumThreads/BlockWidth;
	//second = BlockWidth;//threads per block

	//hipMallocManaged(&Pixels, sizeof(int) * readSize); 
	//hipMallocManaged(&PL, sizeof(pixelLoc) * readSize);

	int frames = Fx;

	std::ifstream fin("d:/data/file_.bin", std::ios::binary);
	fin.read(reinterpret_cast<char*>(h_Pixels), sizeof(int) * readSize);
	fin.close();

	//FILE* file;
	//file = fopen("d:/data/file_name50.txt", "r");
	//if(file == 0)
	//{
	//	printf("bad file name\n");
	//	exit(0);
	//}
	//float temp;
	////	while (!feof (file))
	//for(int i = 0; i < readSize; i++)
	//{  
	//	fscanf(file, "%E", &temp);
	//	h_Pixels[i] = int(temp);
	//	size_file++;
	//}
	int yTotal;

	cout<<"Prior to addition: "<<endl;

	for(int i = 0; i < 10; i++){
		cout<<h_Pixels[i]<<endl;
	}

	//allocate memory space and copy data to device
	HANDLE_ERROR(hipMalloc((void**) &d_Pixels, sizeof(int) * readSize));
	HANDLE_ERROR(hipMalloc((void**) &d_PL, sizeof(pixelLoc) * readSize));
	HANDLE_ERROR(hipMemcpy((void*) d_Pixels, h_Pixels, sizeof(int) * readSize, hipMemcpyHostToDevice));
	//	HANDLE_ERROR(hipMemset((void*) d_PL, 0, sizeof(pixelLoc) * readSize));

	//run kernel for finding Standard Deviation of data
	StdDev<<<gridSize, blockSize>>>(d_Pixels, d_PL, h_Wsize, frames, totalPixs, numProcThds, devThres);

	//rearrange the Loc file for xcorr in next cuda function


	//wait for all to finish and copy data to host
	hipDeviceSynchronize(); 
	code = hipGetLastError();
	if (code != hipSuccess) 
		printf ("Cuda error -- %s\n", hipGetErrorString(code)); 


	HANDLE_ERROR(hipMemcpy(h_PL, d_PL, sizeof(pixelLoc) * readSize, hipMemcpyDeviceToHost));

	//compress list of points, removing points below threshold 
	int j = 0;
	for(int i = 0; i < readSize; i++)
	{
		//if(h_PL[i].sDev < 1 && h_PL[i].win != -1)
		//	{
		//	cout<<"std = "<<h_PL[i].sDev<<"   "<<h_PL[i].win<<endl;
		//	}
		if(h_PL[i].sDev > 0)
		{
			h_PL[j++] = h_PL[i];
		}
	}

	N = j;
	hipFree(d_PL);
	hipFree(d_Pixels);
	HANDLE_ERROR(hipMalloc((void**) &d_PL, sizeof(pixelLoc) * N));

	HANDLE_ERROR(hipMalloc((void**) &d_Pixels, sizeof(int) * readSize));
	int const N1 = N +1;
	unsigned int const corSize = N1*(N1-1)/2;
	PixelxCor *h_Cor;

	//use memory on Host for Kernel not Device due to Size of Array
	HANDLE_ERROR(hipHostAlloc((void**)&h_Cor, sizeof(PixelxCor) * corSize, hipHostMallocMapped));

	//get the address for Kernel write to output array
	HANDLE_ERROR(hipHostGetDevicePointer(&d_Cor, h_Cor, 0));

	//do the regular stuff for passing arrays to Kernel
	HANDLE_ERROR(hipMemcpy((void*) d_Pixels, h_Pixels, sizeof(int) * readSize, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy((void*) d_PL, h_PL, sizeof(pixelLoc) * N, hipMemcpyHostToDevice));

	//int *Indexing = new int[300000];
	//for(int idx = 0; idx < N; idx++)
	//	Indexing[idx] = corSize - ((N1-idx) * (N1-idx - 1))/2; //this needs to be checked

	//now do xcorrelation
	thredMax /= 2;
	int  blocks = (N+thredMax-1)/thredMax;
	if(blocks > gridLimit) blocks = gridLimit;

	XcrossCUDA_same<<<blocks, thredMax, h_Wsize * sizeof(int)>>>(d_Pixels, d_PL,  d_Cor, N1, corSize, h_Wsize);

	hipDeviceSynchronize(); 
	code = hipGetLastError();
	if (code != hipSuccess) 
		printf ("Cuda error -- %s\n", hipGetErrorString(code)); 
	delete[] h_Pixels;
	hipFree(d_Pixels);

	HANDLE_ERROR(hipMemcpy(h_Cor, d_Cor, sizeof(float) * corSize, hipMemcpyDeviceToHost));

	//	int ja = 0;
	//	float *pp = new float[300000];
	//	for(int i = 0, temp = 0; i < corSize; i++)
	//	{
	//		temp = 1000.0*h_Cor[i];
	//		if(abs((int) temp) > 998)
	//		{
	//			pp[ja++] = i;
	////			ja++;
	//		}
	//	}
	//	cout<<"After addition:"<<ja<<endl;
	////create file for Lovain analysis: P1 X Pn....; P2 X Pn-1....
	//for(int i = 0; i < corSize; i++)
	//{
	//	h_final[i].addrss = h_PL[i].win;
	//	h_final[i].Xaddrss = h_PL[i + j].win;
	//	h_final[i].XcorVal = h_Cor[j];
	//}
	//	cout<<ja<<endl;

	//for(int i = 0; i < corSize; i++)
	//	if(int(100 * h_Cor[i].loc_corrCoef) < 1)
	//		continue;
	//write out the data to a file

	FILE *fpw;
	char filew[512];
	sprintf(filew,"%s.pair.txt","cor_weights");
	if ((fpw = fopen(filew,"w"))==NULL)
	{
		printf("cannot open file\n");
	}
	//		printf("\tPt1\t\t\t\Pt2\t\tXcorr\nX\tY\tF\tX\tY\tF\t\n");
	//		fprintf(fpw, "\tPt1\t\t\t\tPt2\t\tXcorr\nX\tY\tF\tX\tY\tF\t\n");
	//		printf("\tPt\#1\tFrm#\t\t\Pt\#2\t\Frm#\tXcorr\n");
	//	fprintf(fpw, "\tPt\#1\tFrm#\t\t\Pt\#2\t\Frm#\tXcorr\n");
	//	fprintf(fpw, "Pt#1\tFrm#\t\Pt#2\t\Frm#\tXcorr\n");
	fprintf(fpw, "Frm#\tPt#1\tPt#2\tXcorr\n");
	for(int i = 0; i < corSize; i++)
	{
		Floc1 = h_Cor[i].loc_Wind1 % frames;
		Floc2 = h_Cor[i].loc_Wind2 % frames;
		Yloc1 = (h_Cor[i].loc_Wind1-Floc1)/frames;
		Yloc2 = (h_Cor[i].loc_Wind2-Floc2)/frames;
		//Xloc1 = Yloc1%imageX;
		//Yloc1 = (Yloc1 - Xloc1)/imageX;
		//Xloc2 = Yloc2%imageX;
		//Yloc2 = (Yloc2 - Xloc2)/imageX;
		//Xloc1 += 1;
		//Xloc2 += 1;
		Yloc1 += 1;
		Yloc2 += 1;
		//if (~Yloc1)
		//	Yloc1=imageX;
		//Xloc2 = floor((h_Cor[i].loc_Wind2-Floc2)/imageY);
		//Yloc2 = (h_Cor[i].loc_Wind2-Floc2) - (Xloc2*imageY);
		if(Floc1 == Floc2)
			fprintf(fpw, "%d\t%d\t%d\t%f\n",Floc1, Yloc1, Yloc2,  h_Cor[i].loc_corrCoef);
		//		fprintf(fpw, "%d\t%d\t%d\t%d\t%f\n",Yloc1, Floc1,Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//if (~Yloc2)
		//	Yloc2=imageX;
		//		fprintf(fpw, "%d\t%d\t%d\t%d\t%f\n",Yloc1, Floc1,Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		printf("%d\t%d\t%d\t%d\t%f\n",Yloc1, Floc1,Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		fprintf(fpw, "%d\t%d\t%d\t%d\t%d\t%d\t%f\n",Xloc1, Yloc1, Floc1, Xloc2, Yloc2, Floc2, h_Cor[i].loc_corrCoef);
		//		fprintf(fpw, "Pt1(x,y,f) = %d,%d,%d Pt2(x,y,f) = %d,%d,%d Xcorr = %f\n",Xloc1, Yloc1, Floc1, Xloc2, Yloc2, Floc2, h_Cor[i].loc_corrCoef);
	}

	fclose(fpw);
	hipHostFree(h_Cor);
	hipFree(d_PL);
	delete[] h_PL;
	return 0;
}
